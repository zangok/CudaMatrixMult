#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hip/hip_bf16.h>
#include <iostream>
#include <hipblaslt.h>
#include <hipblas.h>
#include "cublas_matmul.cuh"
#include "GpuMatrix.h"
#include "custom_matmul_1.cuh"
#include "gemm_policy.h"

// Define function signature for kernels
template <typename T>
using KernelFn = void(*)(int, int, int, T*, T*, T*);

// Kernel registry per type
template <typename T>
std::vector<std::pair<std::string, KernelFn<T>>> get_kernels();

template <>
std::vector<std::pair<std::string, KernelFn<bf16>>> get_kernels<bf16>() {
    return {
        {"cuBLAS bf16 GEMM", runCublasMatmulBF16},
        {"custom bf16 GEMM", runCustomMatmul<GemmPolicyBF16>}
		
    };
}


// Function that runs the various matmul implementations & compares
//N, M, K: Matrix sizes for A(N,M) * B(M,K) = C(N,K)
//iterations: total iterations to average over for each kernel
template <typename T>
void compare_matmul(int N, int M, int K, int iterations) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    GpuMatrix<T> A(N, M);
    GpuMatrix<T> B(M, K);
    GpuMatrix<T> C(N, K);

    A.fill_normal();
    B.fill_normal();

    auto kernels = get_kernels<T>();

    for (auto& kv : kernels) {
        const auto& kernel_name = kv.first;
        auto kernel_fn = kv.second;
        float total_ms = 0.0f;

        for (int i = 0; i < iterations; i++) {
            hipEventRecord(start);

            kernel_fn(N, M, K, A.data(), B.data(), C.data());

            hipEventRecord(stop);
            hipEventSynchronize(stop);

            float ms;
            hipEventElapsedTime(&ms, start, stop);
            total_ms += ms;
        }
        std::cout  << kernel_name << " Average execution time: " << (total_ms / iterations) << " ms\n";
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);
}


int main() {

    initCublasLt();

    compare_matmul<bf16>(4096, 4096, 4096, 1);

}
